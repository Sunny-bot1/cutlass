/***************************************************************************************************
 * Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Example of running an Ada FP8 GEMM.

    D = alpha * accumulator + beta * source
    
*/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gemm.h"

#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/epilogue/thread/linear_combination_generic_with_scaling.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm_splitk_parallel.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/matrix_coord.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/layout/permute.h"


using ElementA = cutlass::float_e4m3_t;
using ElementB = cutlass::float_e4m3_t;
using ElementOutput = cutlass::float_e4m3_t;
using ElementAuxOutput = ElementOutput;
using ElementAccumulator = float;
using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using LayoutC = cutlass::layout::RowMajor;
static int const kStages = 3;
static int const kAlignmentA = 16;
static int const kAlignmentB = 16;

using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput, 
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator>;

template <typename MathOperator>
using Gemm_ = cutlass::gemm::device::GemmSplitKParallel<
    ElementA, LayoutA, 
    ElementB, LayoutB, 
    ElementOutput, LayoutC,
    ElementAccumulator, 
    cutlass::arch::OpClassTensorOp, 
    cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, 
    cutlass::gemm::GemmShape<64, 64, 64>, 
    cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp
    // cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, 
    // kStages,
    // kAlignmentA, 
    // kAlignmentB, 
    // MathOperator
  >;


// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;
  cutlass::gemm::GemmCoord problem_size;

  int iterations;
  int warmup_iterations;
  int split_k = 1;

  float alpha;
  float beta;

  Options():
    help(false),
    error(false),
    reference_check(false),
    iterations(20),
    warmup_iterations(5),
    alpha(1.f),
    beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("warmup_iterations", warmup_iterations, 5);
    cmd.get_cmd_line_argument("reference-check", reference_check, false);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("split_k", split_k, 1);

    int m, n, k;
    cmd.get_cmd_line_argument("m", m, 1280);
    cmd.get_cmd_line_argument("n", n, 4096);
    cmd.get_cmd_line_argument("k", k, 4096);

    problem_size = cutlass::gemm::GemmCoord{m, n, k};
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "58_ada_fp8_gemm\n\n"
      << "  This example executes a GEMM using Ada FP8 Tensor Core operations. In addition to performing\n"
      << "  a normal GEMM, the kernel performs the following operations:\n"
      << "      D = alpha  * accumulator + beta * source \n"
      << "Options:\n\n"
      << "  --help                           If specified, displays this usage statement\n\n"
      << "  --m=<int>                        Sets the M dimension of the GEMM\n"
      << "  --n=<int>                        Sets the N dimension of the GEMM\n"
      << "  --k=<int>                        Sets the K dimension of the GEMM\n"
      << "  --iterations=<int>               Number of profiling iterations to perform\n"
      << "  --warmup-iterations=<int>        Number of warmup iterations to perform\n"
      << "  --reference-check=<bool>         If true, performs reference check\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  float gflops(float runtime_s) const {
    // Two flops per multiply-add
    return 2.0f * float(problem_size.product()) / float(1.0e9) / runtime_s;
  }
};

/// Helper class to run the kernel
template <typename Gemm>
struct TestbedRunner {

  using ElementAccumulator = typename Gemm::ElementAccumulator;
  using ElementCompute = typename Gemm::GemmKernel::Epilogue::OutputOp::ElementCompute;

  /// Initialization
  cutlass::Distribution::Kind init_A;
  cutlass::Distribution::Kind init_B;
  cutlass::Distribution::Kind init_C;
  uint64_t seed;

  cutlass::HostTensor<typename Gemm::ElementA, typename Gemm::LayoutA> tensor_A;
  cutlass::HostTensor<typename Gemm::ElementB, typename Gemm::LayoutB> tensor_B;
  cutlass::HostTensor<typename Gemm::ElementC, typename Gemm::LayoutC> tensor_C;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, typename Gemm::LayoutC> tensor_D;
  cutlass::HostTensor<typename Gemm::ElementC, typename Gemm::LayoutC> tensor_Vector;
  cutlass::HostTensor<ElementAccumulator, typename Gemm::LayoutC> tmp_D;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, typename Gemm::LayoutC> reference_D;
  //
  // Methods
  //

  TestbedRunner(
    bool scaleA = true,
    bool scaleB = true,
    bool scaleC = true,
    cutlass::Distribution::Kind init_A_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_B_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_C_ = cutlass::Distribution::Uniform,
    uint64_t seed_ = 2080
  ):
    init_A(init_A_), init_B(init_B_), init_C(init_C_), seed(seed_) { }

  /// Helper to initialize scaling factors
  template <typename Element, typename Layout>
  bool initialize_scale_factor(cutlass::TensorView<Element, Layout> view, uint64_t seed, int bits=0) {
    cutlass::reference::host::TensorFillRandomUniform(view, seed, double(1.), double(0.), bits);
    return true;
  }

  /// Helper to initialize a tensor view
  template <typename Element, typename Layout>
  bool initialize_tensor(
    cutlass::TensorView<Element, Layout> view,
    cutlass::Distribution::Kind dist_kind,
    uint64_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      double scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<typename Gemm::ElementC>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        scope_max = 5;
        scope_min = -5;
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::host::TensorFillRandomUniform(
        view, seed, scope_max, scope_min, 0);
    }
    else if (dist_kind == cutlass::Distribution::Identity) {

      cutlass::reference::host::TensorFillIdentity(view);
    }
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      cutlass::reference::host::BlockFillSequential(
        view.data(), view.capacity());
    }
    else {
      std::cerr << "Not implemented";
      return false;
    }

    return true;
  }

  /// Initializes data structures
  void initialize(const Options& options) {
    //
    // Allocate the GEMM workspace
    //

    tensor_A.resize(options.problem_size.mk());
    tensor_B.resize(options.problem_size.kn());
    tensor_C.resize(options.problem_size.mn());
    tensor_D.resize(options.problem_size.mn());
    tensor_Vector.resize({1, options.problem_size.n()});
    reference_D.resize(options.problem_size.mn(), false);
    tmp_D.resize(options.problem_size.mn(), false);

    initialize_tensor(tensor_A.host_view(), init_A, seed + 2019);
    initialize_tensor(tensor_B.host_view(), init_B, seed + 2018);
    initialize_tensor(tensor_C.host_view(), init_C, seed + 2017);
    initialize_tensor(tensor_Vector.host_view(), init_C, seed + 2020);

    // It is possible to randomly initialize to all zeros, so override this with non-zeros
    // in the upper left corner of each operand.
    cutlass::Coord<2> origin(0);
    tensor_A.host_view().at(origin) = typename Gemm::ElementA(1);
    tensor_B.host_view().at(origin) = typename Gemm::ElementB(1);
    tensor_C.host_view().at(origin) = typename Gemm::ElementC(1);
    tensor_Vector.host_view().at(origin) = typename Gemm::ElementC(1);

    cutlass::reference::host::TensorFill(tensor_D.host_view());
    cutlass::reference::host::TensorCopy(reference_D.host_view(), tensor_C.host_view());

    tensor_A.sync_device();
    tensor_B.sync_device();
    tensor_C.sync_device();
    tensor_D.sync_device();
    tensor_Vector.sync_device();
  }

  /// Compares computed reference with device reference and outputs to a file if incorrect
  bool compare_reference(const Options& options) {

    tensor_D.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(reference_D.host_view(), tensor_D.host_view());

    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;

      std::string output_file = "testbed_with_amax_errors.txt";
      std::ofstream file(output_file);

      file
        << "problem: " << options.problem_size
        << ", alpha: " << options.alpha << ", beta: " << options.beta << "\n\n";

      file
        << "A =\n" << tensor_A.host_view()
        << "\nB =\n" << tensor_B.host_view()
        << "\nC =\n" << tensor_C.host_view()
        << "\nVector =\n" << tensor_Vector.host_view()
        << "\n\nReference D =\n" << reference_D.host_view()
        << "\nComputed D =\n" << tensor_D.host_view();

      std::cerr << "Dumped results to " << output_file << std::endl;

    }

    return passed;
  }

  /// Verifies the result is a GEMM
  bool verify(const Options& options) {

    cutlass::Coord<2> origin(0);
    ElementCompute scaled_alpha = options.alpha;
    ElementCompute scaled_beta = options.beta;

    //
    // Verify
    //

    cutlass::reference::host::GemmComplex<
        typename Gemm::ElementA, typename Gemm::LayoutA,
        typename Gemm::ElementB, typename Gemm::LayoutB,
        typename Gemm::ElementC, typename Gemm::LayoutC,
        ElementCompute, ElementAccumulator, ElementAccumulator
    >(
      options.problem_size,
      scaled_alpha,
      tensor_A.host_ref(),
      cutlass::ComplexTransform::kNone, 
      tensor_B.host_ref(),
      cutlass::ComplexTransform::kNone,
      scaled_beta,
      tensor_C.host_ref(),
      tmp_D.host_ref(),
      ElementAccumulator(0)
    );

    cutlass::NumericConverter<ElementCompute, typename Gemm::ElementC> cvt_c_to_compute;
    cutlass::NumericConverter<ElementCompute, ElementAccumulator> cvt_accum_to_compute;
    cutlass::NumericConverter<ElementAccumulator, ElementCompute> cvt_compute_to_accum;
    cutlass::NumericConverter<typename Gemm::EpilogueOutputOp::ElementOutput, ElementCompute> cvt_compute_to_d;

    for (int m = 0; m < options.problem_size.m(); ++m) {
      for (int n = 0; n < options.problem_size.n(); ++n) {
        ElementCompute d = cvt_accum_to_compute(tmp_D.host_view().at({m, n}));
        reference_D.host_view().at({m, n}) = cvt_compute_to_d(d);
      }
    }

    return compare_reference(options);
  }

  /// Returns true if the CUDA device is sufficient to execute the kernel.
  bool sufficient() const {

    if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 4)) {
      std::cerr << "This example requires CUDA 12.4 or greater." << std::endl;
      return false;
    }

    size_t smem_size = sizeof(typename Gemm::GemmKernel::SharedStorage);

    hipDeviceProp_t properties;
    int device_idx;
    hipError_t result = hipGetDevice(&device_idx);

    if (result != hipSuccess) {
      std::cerr << "hipGetDevice() failed with error: " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipGetDeviceProperties(&properties, device_idx);

    if (result != hipSuccess) {
      std::cerr << "hipGetDeviceProperties() failed with error: " << hipGetErrorString(result) << std::endl;
      return false;
    }

    if (properties.major < 8 || (properties.major == 8 && properties.minor < 9)) {
      std::cerr << "CUTLASS's Ada FP8 GEMM example requires a device of compute capability 89 or higher.\n" << std::endl;
      return false;
    }

    if (properties.sharedMemPerBlockOptin < smem_size) {
      std::cerr << "Insufficient shared memory. Need " << smem_size
                << ", but device only has " << properties.sharedMemPerBlockOptin << std::endl;
      return false;
    }

    return true;
  }

  /// Executes one test
  bool run(Options& options)
  {

    // Waive test if insufficient CUDA device
    if (!sufficient()) {
      std::cerr << "Insufficient resources to run the kernel." << std::endl;
      return false;
    }

    this->initialize(options);

    typename Gemm::Arguments arguments{
      options.problem_size,
      tensor_A.device_ref(),
      tensor_B.device_ref(),
      tensor_C.device_ref(),
      tensor_D.device_ref(),
      {options.alpha, options.beta},
      options.split_k
    };

    Gemm gemm_op;

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::can_implement() failed" << std::endl;
      return false;
    }

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::initialize() failed" << std::endl;
      return false;
    }

    //
    // Run the GEMM
    //

    status = gemm_op();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::run() failed" << std::endl;
      return false;
    }

    hipError_t cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
      std::cerr << "CUDA error: " << hipGetErrorString(cuda_error) << std::endl;
      return false;
    }

    //
    // Verify
    //

    bool passed = true;
    if (options.reference_check) {
      passed &= this->verify(options);
    } else {
      std::cout << "Skipped reference check" << std::endl;
    }

    //
    // Warm up
    //

    for (int i = 0; i < options.warmup_iterations; ++i) {
      gemm_op();
    }

    //
    // Profile
    //

    hipEvent_t events[2];
    hipError_t error;
    for (auto & event : events) {
      error = hipEventCreate(&event);
      if (error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(error) << std::endl;
        return false;
      }
    }

    // Record an event at the start of a series of GEMM operations
    error = hipEventRecord(events[0]);
    if (error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Run profiling loop
    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm_op();
    }

    // Record an event when the GEMM operations have been launched.
    error = hipEventRecord(events[1]);
    if (error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Wait for work on the device to complete.
    error = hipEventSynchronize(events[1]);
    if (error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Compute average runtime and GFLOPs.
    runtime_ms = runtime_ms / float(options.iterations);
    float gflops = options.gflops(runtime_ms / 1000.0f);

    std::cout << "Problem size: " << options.problem_size.m() << 'x' << options.problem_size.n() << 'x' << options.problem_size.k() << std::endl;
    std::cout << "Runtime (ms): " << runtime_ms << std::endl;
    std::cout << "GFLOPs/sec:   " << gflops << std::endl;

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    return passed;
  }

};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const** argv) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 4) ||
      (props.major != 8 && props.minor != 9)) {

    //
    // This example requires an NVIDIA Ada-architecture GPU.
    //

    std::cout
      << "CUTLASS's FP8 SM89 example requires a GPU of NVIDIA's Ada architecture "
      << "and CUDA toolkit version 12.4 or later.\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  std::cout << "Running GEMM with staged accumulation (OpMultiplyAdd)" << std::endl;
  std::cout << "=====================================================" << std::endl;
  TestbedRunner<Gemm_<cutlass::arch::OpMultiplyAdd>> testbed_staged_accum;
  bool passed = testbed_staged_accum.run(options);

  if (passed) {
    std::cout << "Passed" << std::endl;
  } else {
    std::cout << "Failed" << std::endl;
  }

  std::cout << "\nRunning GEMM with fast accumulation (OpMultiplyAddFastAccum)" << std::endl;
  std::cout << "============================================================" << std::endl;
  TestbedRunner<Gemm_<cutlass::arch::OpMultiplyAddFastAccum>> testbed_fast_accum;
  passed = testbed_fast_accum.run(options);

  if (passed) {
    std::cout << "Passed" << std::endl;
  } else {
    std::cout << "Failed" << std::endl;
  }

  return 0;
}
